#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* d_pixels, int width, float stepX, float stepY, float lowerX, float lowerY, int count) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int index = thisX + thisY * width;

    float c_x = lowerX + thisX * stepX;
    float c_y = lowerY + thisY * stepY;

    float z_x = c_x;
    float z_y = c_y;
    int i; // i: iteration
    for (i = 0; i < count; ++i){
        if (z_x * z_x + z_y * z_y > 4.f) 
            break;

        float new_x = z_x * z_x - z_y * z_y;
        float new_y = 2.f * z_x * z_y;
        z_x = c_x + new_x;
        z_y = c_y + new_y;
    }

    d_pixels[index] = i;    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size=resX * resY *sizeof(int); //N = resX * resY  //size=N*sizeof(int)

    int *h_pixels, *d_pixels;

    // h_pixels = (int *)malloc(size);
    hipMalloc((void **)&d_pixels, size);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlock(resX/threadsPerBlock.x, resY/threadsPerBlock.y);
    
    mandelKernel<<<numBlock, threadsPerBlock>>>(d_pixels, resX, stepX, stepY, lowerX, lowerY, maxIterations);
    
    hipMemcpy(img, d_pixels, size, hipMemcpyDeviceToHost);
    // memcpy(img, h_pixels, size);

    hipFree(d_pixels);
    // free(h_pixels);
}


/*
#include <cuda.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel() {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
}
*/