#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* d_pixels, float stepX, float stepY, float lowerX, float lowerY, int maxIterations, int pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    // int index = thisX + thisY * width;

    float c_x = lowerX + thisX * stepX;
    float c_y = lowerY + thisY * stepY;

    float z_x = c_x;
    float z_y = c_y;
    int i; // i: iteration
    for (i = 0; i < maxIterations; ++i){
        if (z_x * z_x + z_y * z_y > 4.f) 
            break;

        float new_x = z_x * z_x - z_y * z_y;
        float new_y = 2.f * z_x * z_y;
        z_x = c_x + new_x;
        z_y = c_y + new_y;
    }

    int* row = (int*)((char*)d_pixels + thisY * pitch);
    row[thisX] = i;
    // d_pixels[index] = i;    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size=resX * resY *sizeof(int); //N = resX * resY  //size=N*sizeof(int)
    size_t pitch = 0;

    int *h_pixels, *d_pixels;
    hipHostAlloc(&h_pixels, size, hipHostMallocMapped);
    hipMallocPitch(&d_pixels, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlock(resX/threadsPerBlock.x, resY/threadsPerBlock.y);
    
    mandelKernel<<<numBlock, threadsPerBlock>>>(d_pixels, stepX, stepY, lowerX, lowerY, maxIterations,pitch);
    
    hipMemcpy2D(h_pixels,resX * sizeof(int), d_pixels, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_pixels, size);

    hipFree(d_pixels);
    hipHostFree(h_pixels);
}
