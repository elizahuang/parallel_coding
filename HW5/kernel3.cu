#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int* d_pixels, float stepX, float stepY, float lowerX, float lowerY, int maxIterations, int pitch,int pixelsPerThreads) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x)*pixelsPerThreads;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y)*pixelsPerThreads;
    // int index = thisX + thisY * width;

    for (int j = 0; j < pixelsPerThreads; j++){
        for (int i = 0; i < pixelsPerThreads; i++){
            float c_x = lowerX + (thisX + i) * stepX;
            float c_y = lowerY + (thisY + j) * stepY;
            float z_x = c_x;
            float z_y = c_y;

            int iter;
            for (iter = 0; iter < maxIterations; ++iter){
                if (z_x * z_x + z_y * z_y > 4.f) break;

                float new_x = z_x * z_x - z_y * z_y;
                float new_y = 2.f * z_x * z_y;
                z_x = c_x + new_x;
                z_y = c_y + new_y;
            }

            int* row = (int*)((char*)d_pixels + (thisY + j) * pitch);
            row[thisX + i] = iter;
        }
    }   
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size=resX * resY *sizeof(int); //N = resX * resY  //size=N*sizeof(int)
    size_t pitch = 0;
    int pixelsPerThreads = 4;

    int *h_pixels, *d_pixels;
    hipHostAlloc(&h_pixels, size, hipHostMallocMapped);
    hipMallocPitch(&d_pixels, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(25, 25);
    dim3 numBlock(resX/threadsPerBlock.x/pixelsPerThreads, resY/threadsPerBlock.y/pixelsPerThreads);
    
    mandelKernel<<<numBlock, threadsPerBlock>>>(d_pixels, stepX, stepY, lowerX, lowerY, maxIterations,pitch,pixelsPerThreads);
    
    hipMemcpy2D(h_pixels,resX * sizeof(int), d_pixels, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_pixels, size);

    hipFree(d_pixels);
    hipHostFree(h_pixels);
}
